#include <hip/hip_runtime.h>

#include <cute/tensor.hpp>

#include "gemm.cuh"

namespace ct = cute;

int main(int argc, char const *argv[]) {
    if (argc != 5) {
        std::cout << "Usage: " << argv[0] << " M N K iters" << std::endl;
        return 1;
    }
    int64_t M = atoi(argv[1]);
    int64_t N = atoi(argv[2]);
    int64_t K = atoi(argv[3]);
    size_t iters = atoi(argv[4]);

    // Allocate A, B, C
    ct::half_t *A_ptr;
    ct::half_t *B_ptr;
    ct::half_t *C_ptr;
    void *z_ptr;
    hipMalloc(&A_ptr, M * K * sizeof(ct::half_t));
    hipMalloc(&B_ptr, N * K * sizeof(ct::half_t));
    hipMalloc(&C_ptr, M * N * sizeof(ct::half_t));
    hipMalloc(&z_ptr, 3 * 1024 * 1024);  // Size of my L2 cache
    auto A = ct::make_tensor(ct::make_gmem_ptr(A_ptr), ct::make_layout(ct::make_shape(M, K), ct::GenRowMajor{}));
    auto B = ct::make_tensor(ct::make_gmem_ptr(B_ptr), ct::make_layout(ct::make_shape(N, K), ct::GenRowMajor{}));
    auto C = ct::make_tensor(ct::make_gmem_ptr(C_ptr), ct::make_layout(ct::make_shape(M, N), ct::GenRowMajor{}));

    // Time and benchmark
    std::vector<hipEvent_t> start_events;
    std::vector<hipEvent_t> end_events;
    for (size_t i = 0; i < iters; i++) {
        hipEvent_t start_event;
        hipEvent_t end_event;
        hipEventCreate(&start_event);
        hipEventCreate(&end_event);
        start_events.push_back(start_event);
        end_events.push_back(end_event);
    }

    // Start benchmark
    for (size_t i = 0; i < iters; i++) {
        hipMemset(z_ptr, 0, 3 * 1024 * 1024);  // Flush L2 cache
        hipEventRecord(start_events[i]);
        simplegemm::gemm(A, B, C);
        hipEventRecord(end_events[i]);
    }

    // Report benchmark results
    float total_duration = 0;
    for (size_t i = 0; i < iters; i++) {
        hipEventSynchronize(end_events[i]);
        float duration;  // in ms
        hipEventElapsedTime(&duration, start_events[i], end_events[i]);
        total_duration += duration;
    }
    float flops = 2 * M * N * K * iters / (total_duration / 1000);
    float bandwidth = ((M * K) * (N / simplegemm::KernelTraits::BLK_N) + (N * K) * (M / simplegemm::KernelTraits::BLK_M) + M * N) * iters * sizeof(ct::half_t) / (total_duration / 1000);
    std::cout << "Time elapse: " << total_duration << "ms" << std::endl;
    std::cout << "TFLOPS: " << flops / 1e12 << std::endl;
    std::cout << "Bandwidth: " << bandwidth / 1e9 << "GB/s" << std::endl;

    // Deallocate A, B, C
    hipFree(A_ptr);
    hipFree(B_ptr);
    hipFree(C_ptr);
    hipFree(z_ptr);

    return 0;
}