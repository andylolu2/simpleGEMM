#include <hip/hip_runtime.h>

#include <cute/tensor.hpp>

#include "pointwise.cuh"

namespace ct = cute;

int main(int argc, char const *argv[]) {
    if (argc != 4) {
        std::cout << "Usage: " << argv[0] << " M N iters" << std::endl;
        return 1;
    }
    int64_t M = atoi(argv[1]);
    int64_t N = atoi(argv[2]);
    size_t iters = atoi(argv[3]);

    using T = ct::half_t;

    // Allocate A, B, C
    T *A_ptr;
    T *B_ptr;
    hipMalloc(&A_ptr, M * N * sizeof(T));
    hipMalloc(&B_ptr, M * N * sizeof(T));
    auto A = ct::make_tensor(ct::make_gmem_ptr(A_ptr), ct::make_layout(ct::make_shape(M, N), ct::GenRowMajor{}));
    auto B = ct::make_tensor(ct::make_gmem_ptr(B_ptr), ct::make_layout(ct::make_shape(M, N), ct::GenRowMajor{}));

    // Time and benchmark
    std::vector<hipEvent_t> start_events;
    std::vector<hipEvent_t> end_events;
    for (size_t i = 0; i < iters; i++) {
        hipEvent_t start_event;
        hipEvent_t end_event;
        hipEventCreate(&start_event);
        hipEventCreate(&end_event);
        start_events.push_back(start_event);
        end_events.push_back(end_event);
    }

    // Start benchmark
    void *z_ptr;
    hipMalloc(&z_ptr, 3 * 1024 * 1024);  // Size of my L2 cache
    for (size_t i = 0; i < iters; i++) {
        hipMemset(z_ptr, 0, 3 * 1024 * 1024);  // Flush L2 cache
        hipEventRecord(start_events[i]);
        simplegemm::relu(A, B);
        hipEventRecord(end_events[i]);
    }
    hipFree(z_ptr);

    // Report benchmark results
    float total_duration = 0;
    for (size_t i = 0; i < iters; i++) {
        hipEventSynchronize(end_events[i]);
        float duration;  // in ms
        hipEventElapsedTime(&duration, start_events[i], end_events[i]);
        total_duration += duration;
    }
    float tflops = M * N * iters / (total_duration / 1000) / 1E12;
    float bandwidth = 2 * M * N * iters * sizeof(T) / (total_duration / 1000) / 1E9;
    std::cout << "Time elapse: " << total_duration << "ms" << std::endl;
    std::cout << "TFLOPS: " << tflops << std::endl;
    std::cout << "Bandwidth: " << bandwidth << "GB/s" << std::endl;

    // Deallocate A, B, C
    hipFree(A_ptr);
    hipFree(B_ptr);

    return 0;
}